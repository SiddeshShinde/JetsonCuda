#include "hip/hip_runtime.h"
#include<stdio.>
#include<math.h>
#include<string.h>
#include<iostream>
#include<hip/hip_runtime.h>


__global__ void sq(int *d,int n)
{
	__shared__ s[n];
	int t= threadIdx.x;
	if(t==0 || t==2)
		s[t]= d[2*t] * d[2*t+1];
	if(t==1)
		s[t]= 2*(d[2*t]*d[2*t+1]);
	__syncthreads();
	d[2*t]=s[t];
}
int main()
{
	int n=6;
	int ori[6],d[6],ans1;
	int no,x,y;
	
	cout<<"Enter the number";
	cin>>no;
	
	x=(no/10)*10;
	y=no%10;
	
	ori[0]=ori[2]=ori[4]=x;
	ori[1]=ori[3]=ori[6]=y;
	
	int *d_d;
	
	hipMalloc(&d_d,n*sizeof(int));
	hipMemcpy(d_d,ori,n*sizeof(int)),hipMemcpyHostToDevice);
	sq<<<1,n/2>>>(d_d,n/2);
	hipMemcpy(d,d_d,n*sizeof(int)),hipMemcpyDeviceToHost);
	hipFree(d_d);
	
	ans1=d[0]+d[2]+d[4];
	cout<<"Ans is"<<ans1;
	return 0;
}

