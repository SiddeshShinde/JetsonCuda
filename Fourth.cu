
#include <hip/hip_runtime.h>
#include "stdio.h"

#define ROW 3
#define COL 2
__global__ void add(int *a,int *b,int *c)
{
	int x = blockIdx.x;
	int y = blockIdx.x;
	int i = COL*y + x;
	c[i] = a[i] + b[i];
}

int main()
{
	int a[ROW][COL],b[ROW][COL],c[ROW][COL];
	int *dev_a,*dev_b,*dev_c;
	hipMalloc((void**)&dev_a,ROW*COL*sizeof(int));
	hipMalloc((void**)&dev_b,ROW*COL*sizeof(int));
	hipMalloc((void**)&dev_c,ROW*COL*sizeof(int));
	
	for(int y=0;y<ROW;y++)
	{
		for(int x=0;x<COL;x++)
		{
			a[y][x] = x;
			b[y][x] = y;
		}
	}
	
	hipMemcpy(dev_a,a,ROW*COL*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,ROW*COL*sizeof(int),hipMemcpyHostToDevice);
	
	dim3 grid(COL,ROW);
	add<<<grid,1>>>(dev_a,dev_b,dev_c);
	hipMemcpy(c,dev_c,ROW*COL*sizeof(int),hipMemcpyDeviceToHost);

	for(int y=0;y<ROW;y++)
	{
		for(int x=0;x<COL;x++)
		{
			printf("[%d][%d]=%d ",y,x,c[y][x]);
		}
		printf("\n");
	}
	return 0;
}
